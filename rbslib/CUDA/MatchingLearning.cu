#include "hip/hip_runtime.h"
﻿#include "MatchingLearning.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
struct LayerCUDA
{
	float* w;
	int w_rows;
	int w_cols;

	float* b;
	int b_rows;
	int b_cols;

	float* output;
	int output_rows;
	int output_cols;

	float* delta;
	int delta_rows;
	int delta_cols;

	float* z;
	int z_rows;
	int z_cols;

	int activation_func_index;//预先定义好的几个激活函数的下标

};

__global__ void matrix_add_kernel(const float* A, const float* B, float* C, int rows, int cols) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        C[index] = A[index] + B[index];
    }
}

__global__ void matrix_sub_kernel(const float* A, const float* B, float* C, int rows, int cols)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (row < rows && col < cols) {
		int index = row * cols + col;
		C[index] = A[index] - B[index];
	}
}

__global__ void matrix_multiply_kernel(const float* A, const float* B, float* C, int A_rows, int A_cols, int B_cols) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols) {
        float sum = 0.0;
        for (int k = 0; k < A_cols; ++k) {
            sum += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = sum;
    }
}

__global__ void scalar_multiply_kernel(const float* A, float scalar, float* B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;
        B[index] = scalar * A[index];
    }
}

__global__ void matrix_transpose_kernel(const float* A, float* B, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        // 原矩阵 A 的位置 (row, col) 在转置矩阵 B 中变为 (col, row)
        B[col * rows + row] = A[row * cols + col];
    }
}


__device__ float sigmoid(float x) {
    return 1.0 / (1.0 + exp(-x));
}

__device__ float sigmoid_derivative(float x) {
    float sig = sigmoid(x);
    return sig * (1.0 - sig);
}

__global__ void apply_function_kernel(float* matrix, int rows, int cols, int activation_func_index) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int index = row * cols + col;

        if (activation_func_index == 1) {
            // 应用 Sigmoid 激活函数
            matrix[index] = sigmoid(matrix[index]);
        }
        else if (activation_func_index==-1)
        {
            //对应的导数
			matrix[index] = sigmoid_derivative(matrix[index]);
        }
        // 可以在此处添加更多的激活函数，例如 Tanh、ReLU 等
    }
}

__global__ void transpose_multiply_kernel(
    const float* X,       // 输入矩阵 X (m x n)
    const float* Y,       // 输入矩阵 Y (n x p)
    float* Z,             // 输出矩阵 Z (n x p)
    int m,                 // X 的行数
    int n,                 // X 的列数，Y 的行数
    int p                  // Y 的列数
) {
    // 计算 Z 矩阵的元素位置 (i, j)
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // i 表示 Z 的行
    int j = blockIdx.y * blockDim.y + threadIdx.y;  // j 表示 Z 的列

    if (i < n && j < p) {
        float sum = 0.0;
        for (int k = 0; k < m; ++k) {
			//sum += X[k][i] * Y[k][j];
            sum += X[k * n + i] * Y[k * p + j];  // X^T * Y 的元素计算
        }
        Z[i * p + j] = sum;
    }
}


// 核函数：逐元素相乘
__global__ void matrix_elementwise_multiply_kernel(const float* A, const float* B, float* C, int rows, int cols) {
    // 计算全局线程 ID
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // 计算对应元素
    if (row < rows && col < cols) {
        int index = row * cols + col; // 计算矩阵元素的线性索引
        C[index] = A[index] * B[index];
    }
}

//前向传播 A[l] = W * A[l-1] + b
__global__ void forward_kernel(const float* W, const float* A, const float* b, float* Z, float* OUT, int W_rows, int W_cols, int act_index)
{
	//结果为单行或单列向量，直接展平计算
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < W_rows)
	{
		float sum = 0;
		for (int k = 0; k < W_cols; ++k)
		{
			sum += W[row * W_cols + k] * A[k];
		}
		Z[row] = sum + b[row];
		if (act_index == 1)
		{
			OUT[row] = sigmoid(Z[row]);
		}
	}
}

/*最后一层delta计算*/
__global__ void backward_last_kernel(const float* output, const float* target, float* delta, float* z, int rows, int act_index,const float* A)
{
	//均为单行或单列向量，直接展平计算
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < rows)
	{
		if (act_index == 1)
		{
			//delta[row] = (output[row] - target[row]) * sigmoid_derivative(z[row]);
			delta[row] = (output[row] - target[row]) * A[row] * (1 - A[row]);
		}
	}
}
//δ[L] = W[L + 1] ^ T * δ[L + 1].*f'(Z[L])
__global__ void backward_kernel(const float* W, const float* delta, float* z, float* next_delta, int W_rows, int W_cols, int act_index,const float* A)
{
	//目标矩阵为单列向量，直接x展平计算
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < W_cols)
	{
		float sum = 0;
		for (int k = 0; k < W_rows; ++k)
		{
			sum += W[k * W_cols + row] * delta[k];
		}
		if (act_index == 1)
		{
			//next_delta[row] = sum * sigmoid_derivative(z[row]);
			//f'(Z[row])可以用A[row]*(1-A[row])表示
			next_delta[row] = sum * A[row] * (1 - A[row]);
		}
	}
}
//W[l] = W[l] - η * δ[l] * A[l-1]^T
//b[l] = b[l] - η * δ[l]
__global__ void update_w_kernel(float* W, int W_row, int W_col, float learn, const float* delta, const float* AT)
{
	//二维数组计算，不展平
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < W_row && col < W_col)
	{
		W[row * W_col + col] -= learn * delta[row] * AT[col];
	}
}

__global__ void update_b_kernel(float* b, int b_row, int b_col, float learn, const float* delta)
{
	//单列向量，直接展平计算
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < b_row)
	{
		b[row] -= learn * delta[row];
	}
}

__global__ void add_loss_kernel(float* A, float* target, float* loss, int rows)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < rows)
	{
		loss[row] += 0.5 * (A[row] - target[row]) * (A[row] - target[row]);
	}
}


void matrix_add(const float* A, const float* B, float* C, int rows, int cols)
{
	//已经确保A B C在GPU上
	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	matrix_add_kernel <<<grid, block >>> (A, B, C, rows, cols);
}

void matrix_sub(const float* A, const float* B, float* C, int rows, int cols)
{
	//已经确保A B C在GPU上
	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	matrix_sub_kernel <<<grid, block >>> (A, B, C, rows, cols);
}

void matrix_multiply(const float* A, const float* B, float* C, int A_rows, int A_cols, int B_cols)
{
    //已经确保A B C在GPU上
    dim3 block(32, 32);
    dim3 grid((B_cols + block.x - 1) / block.x, (A_rows + block.y - 1) / block.y);
    matrix_multiply_kernel <<<grid, block >>> (A, B, C, A_rows, A_cols, B_cols);
}

void scalar_multiply(const float* A, float scalar, float* B, int rows, int cols)
{
	//已经确保A B在GPU上
	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	scalar_multiply_kernel <<<grid, block >>> (A, scalar, B, rows, cols);
}

void matrix_transpose(const float* A, float* B, int rows, int cols)
{
	//已经确保A B在GPU上
	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	matrix_transpose_kernel <<<grid, block >>> (A, B, rows, cols);
}
void matrix_apply_function(float* matrix, int rows, int cols, int activation_func_index)
{
	//已经确保matrix在GPU上
	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	apply_function_kernel <<<grid, block >>> (matrix, rows, cols, activation_func_index);
}

void matrix_elementwise_multiply(const float* A, const float* B, float* C, int rows, int cols)
{
	//A B C已经在GPU上
	dim3 block(32, 32);
	dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
	matrix_elementwise_multiply_kernel <<<grid, block >>> (A, B, C, rows, cols);
}
//用于计算X^T*Y
void transpose_multiply(const float* X, const float* Y, float* Z, int m, int n, int p)
{
	//已经确保X Y Z在GPU上
	dim3 block(32, 32);
	dim3 grid((n + block.x - 1) / block.x, (p + block.y - 1) / block.y);
	transpose_multiply_kernel <<<grid, block >>> (X, Y, Z, m, n, p);
}

void forward(const float* W, const float* A, const float* b, float* Z, float* OUT, int rows, int cols, int act_index)
{
	//已经确保W A b Z OUT在GPU上
	forward_kernel <<<(rows + 1024 - 1) / 1024, 1024 >>> (W, A, b, Z, OUT, rows, cols, act_index);
}

void backward_last(const float* output, const float* target, float* delta, float* z, int rows, int act_index,const float* A)
{
	//已经确保output target delta z在GPU上
	backward_last_kernel <<<(rows + 1024 - 1) / 1024, 1024 >> > (output, target, delta, z, rows, act_index,A);
}

void backward(const float* W, const float* delta, float* z, float* next_delta, int W_rows, int W_cols, int act_index,const float* A)
{
	//已经确保W delta z next_delta在GPU上
	backward_kernel <<<(W_rows + 1024 - 1) / 1024, 1024 >>> (W, delta, z, next_delta, W_rows, W_cols,  act_index,A);
}

void update_w(float* W, int W_row, int W_col, float learn, const float* delta, const float* AT)
{
	//已经确保W delta AT在GPU上
	dim3 block(32, 32);
	dim3 grid((W_col + block.x - 1) / block.x, (W_row + block.y - 1) / block.y);
	update_w_kernel << <grid, block >> > (W, W_row, W_col, learn, delta, AT);
}

void update_b(float* b, int b_row, int b_col, float learn, const float* delta)
{
	//已经确保b delta在GPU上
	update_b_kernel <<<(b_row + 1024 - 1) / 1024, 1024 >>> (b, b_row, b_col, learn, delta);
}

void add_loss(float* A, float* target, float* loss, int rows)
{
	//已经确保A target loss在GPU上
	if (rows<=512)
		add_loss_kernel << <(rows + 512 - 1) / 512, 512 >> > (A, target, loss, rows);
	else 
		add_loss_kernel << <(rows + 1024 - 1) / 1024, 1024 >> > (A, target, loss, rows);
}

void __PrintGPUMatrix(float* matrix, int rows, int cols)
{
	using namespace std;
	RbsLib::Math::Matrix<float> mat(rows, cols);
	hipMemcpy(mat.Data(), matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
	cout << endl << mat.ToString() << endl;
}


void __TrainCUDA(RbsLib::Math::Matrix<float> inputs, RbsLib::Math::Matrix<float> target, float learning_rate, int epochs, std::function<void(int, float)> loss_callback, std::vector<RbsLib::MatchingLearning::Layer>& layers, int activite_index)
{
	//将layers转化为LayerCUDA
	std::vector<LayerCUDA> layers_cuda;
	for (int i = 0; i < layers.size(); i++)
	{
		LayerCUDA layer_cuda;
		layer_cuda.w = layers[i].w.Data();
		layer_cuda.w_rows = layers[i].w.Rows();
		layer_cuda.w_cols = layers[i].w.Cols();

		layer_cuda.b = layers[i].b.Data();
		layer_cuda.b_rows = layers[i].b.Rows();
		layer_cuda.b_cols = layers[i].b.Cols();

		layer_cuda.output = layers[i].output.Data();
		layer_cuda.output_rows = layers[i].output.Rows();
		layer_cuda.output_cols = layers[i].output.Cols();

		layer_cuda.delta = layers[i].delta.Data();
		layer_cuda.delta_rows = layers[i].delta.Rows();
		layer_cuda.delta_cols = layers[i].delta.Cols();

		layer_cuda.z = layers[i].z.Data();
		layer_cuda.z_rows = layers[i].z.Rows();
		layer_cuda.z_cols = layers[i].z.Cols();

		layer_cuda.activation_func_index = activite_index;

		layers_cuda.push_back(layer_cuda);
	}
	//将LayerCUDA中的数组拷贝到GPU，并将指针指向GPU
	int n = 0;
	for (auto& it : layers_cuda)
	{
		hipMalloc(&it.w, it.w_rows * it.w_cols * sizeof(float));
		hipMemcpy(it.w, layers[n].w.Data(), it.w_rows * it.w_cols * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc(&it.b, it.b_rows * it.b_cols * sizeof(float));
		hipMemcpy(it.b, layers[n].b.Data(), it.b_rows * it.b_cols * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc(&it.output, it.output_rows * it.output_cols * sizeof(float));
		//hipMemcpy(it.output, layers[n].output.Data(), it.output_rows * it.output_cols * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc(&it.delta, it.delta_rows * it.delta_cols * sizeof(float));
		//hipMemcpy(it.delta, layers[n].delta.Data(), it.delta_rows * it.delta_cols * sizeof(float), hipMemcpyHostToDevice);

		hipMalloc(&it.z, it.z_rows * it.z_cols * sizeof(float));
		//hipMemcpy(it.z, layers[n].z.Data(), it.z_rows * it.z_cols * sizeof(float), hipMemcpyHostToDevice);
		++n;
	}
    //在GPU上申请空间用于反向传播时存放target
	float* target_cuda;
	float* lossp;
	hipMalloc(&target_cuda, target.Rows() * target.Cols() * sizeof(float));
	hipMemcpy(target_cuda, target.Data(), target.Rows() * target.Cols() * sizeof(float), hipMemcpyHostToDevice);
	//在GPU上申请空间用于存放loss
	hipMalloc(&lossp, sizeof(float)*target.Rows());
	float* lossh = new float[target.Rows()];
	//将主机上的lossh拷贝到GPU(初始化为0)
	hipMemcpy(lossp, lossh, sizeof(float) * target.Rows(), hipMemcpyHostToDevice);
	//将输入拷贝到GPU
	float* inputs_cuda;
	hipMalloc(&inputs_cuda, inputs.Rows() * inputs.Cols() * sizeof(float));
	hipMemcpy(inputs_cuda, inputs.Data(), inputs.Rows() * inputs.Cols() * sizeof(float), hipMemcpyHostToDevice);


    // 开始训练
    for (int e = 0; e < epochs; ++e)
    {
        // 第 e 轮
        float total_loss = 0.0;  // 用来累计损失
        for (int i = 0; i < inputs.Rows(); ++i)
        {
			int index = i; rand() % inputs.Rows();  // 随机选择一个样本

            // 前向传播
            // 第 1 层的输出就是输入
			//将输入拷贝到GPU第一层的输出
			hipMemcpyAsync(layers_cuda[0].output, inputs_cuda + index * inputs.Cols(), inputs.Cols() * sizeof(float), hipMemcpyDeviceToDevice,0);

            // 从第二层开始
            for (int j = 1; j < layers.size(); ++j)
            {
				/*
                // 在GPU中计算当前层的输出 Z = W * A + b
				// Z = W * A
				//检查Z的内容
				matrix_multiply(layers_cuda[j].w, layers_cuda[j - 1].output, layers_cuda[j].z, layers_cuda[j].w_rows, layers_cuda[j].w_cols, layers_cuda[j - 1].output_cols);
                //hipDeviceSynchronize();

				// Z = Z + b
				matrix_add(layers_cuda[j].z, layers_cuda[j].b, layers_cuda[j].z, layers_cuda[j].z_rows, layers_cuda[j].z_cols);
                //hipDeviceSynchronize();
				//将Z拷贝到output
				
				hipMemcpyAsync(layers_cuda[j].output, layers_cuda[j].z, layers_cuda[j].z_rows * layers_cuda[j].z_cols * sizeof(float), hipMemcpyDeviceToDevice, 0);

                // 应用激活函数
				matrix_apply_function(layers_cuda[j].output, layers_cuda[j].output_rows, layers_cuda[j].output_cols, layers_cuda[j].activation_func_index);
                //hipDeviceSynchronize();
				*/
				forward(layers_cuda[j].w, layers_cuda[j - 1].output, layers_cuda[j].b, layers_cuda[j].z, layers_cuda[j].output, layers_cuda[j].w_rows, layers_cuda[j].w_cols, layers_cuda[j].activation_func_index);
            }

            // 计算损失（均方误差）
			//将结果拷贝到CPU
			add_loss(layers_cuda.back().output, target_cuda + index * target.Cols(), lossp, target.Cols());

            // 反向传播
			/*
			// 计算输出层的误差项 δ[L] = ∇C ⊙ f'(Z[L])
			//target已经在GPU上
			//delta = output - target
			matrix_sub(layers_cuda.back().output, target_cuda+index*target.Cols() , layers_cuda.back().delta, layers_cuda.back().delta_rows, layers_cuda.back().delta_cols);
			// Z = f'(Z)
			matrix_apply_function(layers_cuda.back().z, layers_cuda.back().z_rows, layers_cuda.back().z_cols, -layers_cuda.back().activation_func_index);
            //hipDeviceSynchronize();
			//delta = delta .* Z
			matrix_elementwise_multiply(layers_cuda.back().delta, layers_cuda.back().z, layers_cuda.back().delta, layers_cuda.back().delta_rows, layers_cuda.back().delta_cols);
			*/
			backward_last(layers_cuda.back().output, target_cuda + index * target.Cols(), layers_cuda.back().delta, layers_cuda.back().z, layers_cuda.back().delta_rows, layers_cuda.back().activation_func_index,layers_cuda.back().output);
            // 计算隐藏层的误差项 δ[L-1], δ[L-2], ..., δ[1]
			// δ[L] = W[L+1]^T * δ[L+1] .* f'(Z[L])
            for (int j = layers_cuda.size() - 2; j > 0; --j)
            {
				/*
				// δ[l] = W[l+1]^T * δ[l+1]
				transpose_multiply(layers_cuda[j + 1].w, layers_cuda[j + 1].delta, layers_cuda[j].delta, layers_cuda[j + 1].w_rows, layers_cuda[j + 1].w_cols, layers_cuda[j + 1].delta_cols);
				
				// Z[l] = f'(Z[l])
				matrix_apply_function(layers_cuda[j].z, layers_cuda[j].z_rows, layers_cuda[j].z_cols, -layers_cuda[j].activation_func_index);
				// δ[l] = δ[l] .* Z[l]
				matrix_elementwise_multiply(layers_cuda[j].delta, layers_cuda[j].z, layers_cuda[j].delta, layers_cuda[j].delta_rows, layers_cuda[j].delta_cols);
				*/
				backward(layers_cuda[j + 1].w, layers_cuda[j + 1].delta, layers_cuda[j].z, layers_cuda[j].delta, layers_cuda[j + 1].w_rows, layers_cuda[j + 1].w_cols, layers_cuda[j].activation_func_index, layers_cuda[j].output);
            }
            

            // 更新权重和偏置
            for (int l = 1; l < layers.size(); ++l)
            {
				/*
				// W[l] = W[l] - η * δ[l] * A[l-1]^T
				//temp = delta[l] * A[l-1]^T A是一维向量，其转置在内存中的结构是一样的，因此不需要转置
				float* temp;
				hipMallocAsync(&temp, layers_cuda[l].w_rows * layers_cuda[l].w_cols * sizeof(float), 0);
				matrix_multiply(layers_cuda[l].delta, layers_cuda[l - 1].output, temp, layers_cuda[l].delta_rows, layers_cuda[l].delta_cols, 1);//转置只需要将原数组的行作为列即可
				//temp = -η * temp
				scalar_multiply(temp, -learning_rate, temp, layers_cuda[l].delta_rows, layers_cuda[l].delta_cols);

				//w = w + temp
				matrix_add(layers_cuda[l].w, temp, layers_cuda[l].w, layers_cuda[l].w_rows, layers_cuda[l].w_cols);
				hipFreeAsync(temp,0);

				//b[l] = b[l] - η * δ[l] 这是最后一次使用delta，因此不需要再次申请空间
				scalar_multiply(layers_cuda[l].delta, -learning_rate, layers_cuda[l].delta, layers_cuda[l].delta_rows, layers_cuda[l].delta_cols);
				matrix_add(layers_cuda[l].b, layers_cuda[l].delta, layers_cuda[l].b, layers_cuda[l].b_rows, layers_cuda[l].b_cols);
				//输出权值矩阵
				*/
				update_w(layers_cuda[l].w, layers_cuda[l].w_rows, layers_cuda[l].w_cols, learning_rate, layers_cuda[l].delta, layers_cuda[l - 1].output);
				update_b(layers_cuda[l].b, layers_cuda[l].b_rows, layers_cuda[l].b_cols, learning_rate, layers_cuda[l].delta);
            }
        }
        // 输出每轮的损失
		//将loss拷贝到CPU
		hipMemcpyAsync(lossh, lossp, sizeof(float) * target.Rows(), hipMemcpyDeviceToHost,0);
		for (int i = 0; i < target.Rows(); i++) total_loss += lossh[i];
		//重置loss
		std::memset(lossh, 0, sizeof(float) * target.Rows());
		hipMemcpyAsync(lossp, lossh, sizeof(float) * target.Rows(), hipMemcpyHostToDevice,0);
        loss_callback(e, total_loss / inputs.Rows());
    }
	delete[] lossh;
	//将结果拷贝回Layers
	hipDeviceSynchronize();
	for (int i = 0; i < layers.size(); i++)
	{
		hipMemcpy(layers[i].w.Data(), layers_cuda[i].w, layers_cuda[i].w_rows * layers_cuda[i].w_cols * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(layers[i].b.Data(), layers_cuda[i].b, layers_cuda[i].b_rows * layers_cuda[i].b_cols * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(layers[i].output.Data(), layers_cuda[i].output, layers_cuda[i].output_rows * layers_cuda[i].output_cols * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(layers[i].delta.Data(), layers_cuda[i].delta, layers_cuda[i].delta_rows * layers_cuda[i].delta_cols * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(layers[i].z.Data(), layers_cuda[i].z, layers_cuda[i].z_rows * layers_cuda[i].z_cols * sizeof(float), hipMemcpyDeviceToHost);
	}

	//释放GPU空间
	for (auto& it : layers_cuda)
	{
		hipFree(it.w);
		hipFree(it.b);
		hipFree(it.output);
		hipFree(it.delta);
		hipFree(it.z);
	}
	hipFree(target_cuda);
	hipFree(lossp);
	hipFree(inputs_cuda);
}